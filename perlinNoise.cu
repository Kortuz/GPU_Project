#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <random>

#include <math.h>
#include <glm/glm.hpp>
#include <iostream>

# define M_PI 3.14159265358979323846  /* pi */

std::random_device rd; // obtain a random number from hardware
std::mt19937 gen(rd()); // seed the generator
std::uniform_real_distribution<> distr(-1, 1);

__device__ float fade(float t) {
    return t*t*(3.0-2.0*t); // Old fade
    //return t * t * t * (t * (t * 6.0 - 15.0) + 10.0); // Improved fade
}

__global__ void perlinNoise(glm::vec2 *gradGrid, float *heightMap, int gradWidth, int gradHeight, int hmapWidth, int hmapHeight, float frequency, float amplitude)
{
    int gradSize = (gradWidth * gradHeight);

    float hx = ((threadIdx.x + blockDim.x * blockIdx.x) % hmapWidth);
    float hy = ((threadIdx.x + blockDim.x * blockIdx.x) / hmapWidth);
    
    glm::vec2 p = glm::vec2(frequency * (float) hx / (float) gradWidth, frequency * (float) hy / (float) gradHeight);

    glm::vec2 w = glm::vec2(p.x - floor(p.x), p.y - floor(p.y));

    glm::vec2 downLeft = glm::vec2(floor(p.x), floor(p.y));
    glm::vec2 downRight = glm::vec2(downLeft.x + 1.0f, downLeft.y + 0.0f);
    glm::vec2 upLeft = glm::vec2(downLeft.x + 0.0f, downLeft.y + 1.0f);
    glm::vec2 upRight = glm::vec2(downLeft.x + 1.0f, downLeft.y + 1.0f);

    glm::vec2 g0 = gradGrid[((int)(downLeft.x + gradWidth * downLeft.y) % gradSize)];
    glm::vec2 g1 = gradGrid[((int)(downRight.x + gradWidth * downRight.y) % gradSize)];
    glm::vec2 g2 = gradGrid[((int)(upLeft.x + gradWidth * upLeft.y) % gradSize)];
    glm::vec2 g3 = gradGrid[((int)(upRight.x + gradWidth * upRight.y) % gradSize)];

    float up = (1.0 - fade(w.x)) * dot(g0, glm::vec2(p.x - downLeft.x, p.y - downLeft.y)) + fade(w.x) * dot(g1, glm::vec2(p.x - downRight.x, p.y - downRight.y));
    float down = (1.0 - fade(w.x)) * dot(g2, glm::vec2(p.x - upLeft.x, p.y - upLeft.y)) + fade(w.x) * dot(g3, glm::vec2(p.x - upRight.x, p.y - upRight.y));

    heightMap[(threadIdx.x + blockDim.x * blockIdx.x)] += ((1.0 - fade(w.y)) * up + fade(w.y) * down) * amplitude;


}

float* runPerlinNoise(glm::vec2 *gradGrid, float *heightMap, int noiseW, int noiseH, int gradW, int gradH)
{

    float *d_hMap;
    glm::vec2 *d_gGrid;

    for (int i = 0; i < gradW * gradH; i++)
    {
        gradGrid[i].x = distr(gen);
        gradGrid[i].y = distr(gen);
    }

    hipMalloc(&d_hMap, sizeof(float) * noiseW * noiseH);
    hipMalloc(&d_gGrid, sizeof(glm::vec2) * gradW * gradH);

    hipMemcpy(d_gGrid, gradGrid, gradW * gradH * sizeof(glm::vec2), hipMemcpyHostToDevice);
    int N = noiseH * noiseW;

    // Run kernel for each octave
    for (int i = 0; i < 10; i++)
    {
        perlinNoise << <(N + 255) / 256, 256 >> > (d_gGrid, d_hMap, noiseW, noiseH, gradW, gradH, pow(2.0f, i), pow(0.99f, i));
        hipDeviceSynchronize();
        hipMemcpy(heightMap, d_hMap, noiseW * noiseH * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(d_hMap);
    hipFree(d_gGrid);

    return heightMap;
}

__global__ void interpolate(float* d_hMT, float* hM1, float* hM2, float* hM3, float dt)
{
    float w1 = cos((M_PI / 3) * (dt + 2));
    float w2 = cos((M_PI / 3) * (dt + 1));
    float w3 = cos((M_PI / 3) * (dt));

    float inter = (w1 * hM1[(threadIdx.x + blockDim.x * blockIdx.x)]) + (w2 * hM2[(threadIdx.x + blockDim.x * blockIdx.x)]) + (w3 * hM3[(threadIdx.x + blockDim.x * blockIdx.x)]);

    d_hMT[(threadIdx.x + blockDim.x * blockIdx.x)] = inter;
}

float* runInterpolate(float* hMT, float* hM1, float* hM2, float* hM3, int noiseSize, float dt)
{
    float* d_hMT;
    float* d_hM1;
    float* d_hM2;
    float* d_hM3;

    hipMalloc(&d_hMT, sizeof(float) * noiseSize);
    hipMalloc(&d_hM1, sizeof(float) * noiseSize);
    hipMalloc(&d_hM2, sizeof(float) * noiseSize);
    hipMalloc(&d_hM3, sizeof(float) * noiseSize);

    hipMemcpy(d_hM1, hM1, sizeof(float) * noiseSize, hipMemcpyHostToDevice);
    hipMemcpy(d_hM2, hM2, sizeof(float) * noiseSize, hipMemcpyHostToDevice);
    hipMemcpy(d_hM3, hM3, sizeof(float) * noiseSize, hipMemcpyHostToDevice);

    interpolate << <(noiseSize + 255) / 256, 256 >> > (d_hMT, d_hM1, d_hM2, d_hM3, dt);

    hipMemcpy(hMT, d_hMT, sizeof(float) * noiseSize, hipMemcpyDeviceToHost);

    hipFree(d_hMT);
    hipFree(d_hM1);
    hipFree(d_hM2);
    hipFree(d_hM3);
    return hMT;
}